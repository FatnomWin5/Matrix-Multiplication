﻿#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <cstdlib>

using namespace std;

__global__ void matrixMult(int N, const int* matrix_one, const int* matrix_two, int* matrix_res) {
    int i = N * (blockDim.y * blockIdx.y + threadIdx.y);
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int sum = 0;

    for (int k = 0; k < N; k++){
        sum += matrix_one[i + k] * matrix_two[k * N + j];
    }

    int ind = N * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
    matrix_res[ind] = sum;
}

void randomiseMatrix(int* matrix, int N) {

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            matrix[i * N + j] = rand() % 100;
        }
    }
    return;
}

int main(int argc, char** argv) {

    int N;
    int threads;

    printf("Enter matrix size (integer value): ");
    cin >> N;

    printf("Enter max number of threads for grid block (integer value): ");
    cin >> threads;

    int* matrix_one;
    int* matrix_two;
    int* matrix_res;

    size_t size = N * N * sizeof(int);

    matrix_one = (int*)malloc(size);
    matrix_two = (int*)malloc(size);
    matrix_res = (int*)malloc(size);

    randomiseMatrix(matrix_one, N);
    randomiseMatrix(matrix_two, N);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int* calcMatrix_one = NULL;
    int* calcMatrix_two = NULL;
    int* calcMatrix_res = NULL;

    hipMalloc((void**)&calcMatrix_one, size);
    hipMalloc((void**)&calcMatrix_two, size);
    hipMalloc((void**)&calcMatrix_res, size);

    hipMemcpy(calcMatrix_one, matrix_one, size, hipMemcpyHostToDevice);
    hipMemcpy(calcMatrix_two, matrix_two, size, hipMemcpyHostToDevice);

    float time;

    for (int thr = 1; thr <= threads; thr++) {
        dim3 threadsPerBlock = dim3(thr, thr);
        dim3 blocksPerGrid = dim3(N / thr, N / thr);

        hipEventRecord(start, 0);
        matrixMult <<<blocksPerGrid, threadsPerBlock>>> (N, calcMatrix_one, calcMatrix_two, calcMatrix_res);
        hipEventRecord(stop, 0);

        hipEventSynchronize(stop);
        
        hipEventElapsedTime(&time, start, stop);

        printf("Number of threads for grid block: %d; Blocks per grid: %d; Number of seconds: %f", thr, (N / thr), (time / 1000));
        printf("\n");

        hipMemcpy(matrix_res, calcMatrix_res, size, hipMemcpyDeviceToHost);
    }

    hipFree(calcMatrix_one);
    hipFree(calcMatrix_two);
    hipFree(calcMatrix_res);
    free(matrix_one);
    free(matrix_two);
    free(matrix_res);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
